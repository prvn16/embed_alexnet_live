//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: alexnet_predict_terminate.cu
//
// GPU Coder version                    : 1.0
// CUDA/C/C++ source code generated on  : 23-Jan-2018 09:38:17
//

// Include Files
#include "rt_nonfinite.h"
#include "alexnet_predict.h"
#include "alexnet_predict_terminate.h"

// Function Definitions

//
// Arguments    : void
// Return Type  : void
//
void alexnet_predict_terminate()
{
  // (no terminate code required)
}

//
// File trailer for alexnet_predict_terminate.cu
//
// [EOF]
//
