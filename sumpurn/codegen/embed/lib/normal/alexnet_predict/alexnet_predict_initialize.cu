//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: alexnet_predict_initialize.cu
//
// GPU Coder version                    : 1.0
// CUDA/C/C++ source code generated on  : 23-Jan-2018 09:38:17
//

// Include Files
#include "rt_nonfinite.h"
#include "alexnet_predict.h"
#include "alexnet_predict_initialize.h"

// Function Definitions

//
// Arguments    : void
// Return Type  : void
//
void alexnet_predict_initialize()
{
  rt_InitInfAndNaN(8U);
  alexnet_predict_init();
}

//
// File trailer for alexnet_predict_initialize.cu
//
// [EOF]
//
