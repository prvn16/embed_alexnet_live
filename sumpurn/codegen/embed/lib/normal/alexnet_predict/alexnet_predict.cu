//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: alexnet_predict.cu
//
// GPU Coder version                    : 1.0
// CUDA/C/C++ source code generated on  : 23-Jan-2018 09:38:17
//

// Include Files
#include "rt_nonfinite.h"
#include "alexnet_predict.h"
#include "DeepLearningNetwork.h"

// Variable Definitions
static b_alexnet mynet;
static boolean_T mynet_not_empty;

// Function Definitions

//
// A persistent object mynet is used to load the series network object.
//  At the first call to this function, the persistent object is constructed and
//  setup. When the function is called subsequent times, the same object is reused
//  to call predict on inputs, thus avoiding reconstructing and reloading the
//  network object.
// Arguments    : const real_T in[154587]
//                real32_T out[1000]
// Return Type  : void
//
void alexnet_predict(const real_T in[154587], real32_T out[1000])
{
  //  Copyright 2017 The MathWorks, Inc.
  if (!mynet_not_empty) {
    DeepLearningNetwork_setup(&mynet);
    mynet_not_empty = true;
  }

  //  pass in input
  DeepLearningNetwork_predict(&mynet, in, out);
}

//
// Arguments    : void
// Return Type  : void
//
void alexnet_predict_init()
{
  mynet_not_empty = false;
}

//
// File trailer for alexnet_predict.cu
//
// [EOF]
//
