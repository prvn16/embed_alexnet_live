//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: main.cu
//
// GPU Coder version                    : 1.0
// CUDA/C/C++ source code generated on  : 23-Jan-2018 09:38:17
//

//***********************************************************************
// This automatically generated example C main file shows how to call
// entry-point functions that MATLAB Coder generated. You must customize
// this file for your application. Do not modify this file directly.
// Instead, make a copy of this file, modify it, and integrate it into
// your development environment.
//
// This file initializes entry-point function arguments to a default
// size and value before calling the entry-point functions. It does
// not store or use any values returned from the entry-point functions.
// If necessary, it does pre-allocate memory for returned values.
// You can use this file as a starting point for a main function that
// you can deploy in your application.
//
// After you copy the file, and before you deploy it, you must make the
// following changes:
// * For variable-size function arguments, change the example sizes to
// the sizes that your application requires.
// * Change the example values of function arguments to the values that
// your application requires.
// * If the entry-point functions return values, store these values or
// otherwise use them as required by your application.
//
//***********************************************************************
// Include Files
#include "rt_nonfinite.h"
#include "alexnet_predict.h"
#include "main.h"
#include "alexnet_predict_terminate.h"
#include "alexnet_predict_initialize.h"

// Function Declarations
static void argInit_227x227x3_real_T(real_T result[154587]);
static real_T argInit_real_T();
static void main_alexnet_predict();

// Function Definitions

//
// Arguments    : real_T result[154587]
// Return Type  : void
//
static void argInit_227x227x3_real_T(real_T result[154587])
{
  int32_T idx0;
  int32_T idx1;
  int32_T idx2;

  // Loop over the array to initialize each element.
  for (idx0 = 0; idx0 < 227; idx0++) {
    for (idx1 = 0; idx1 < 227; idx1++) {
      for (idx2 = 0; idx2 < 3; idx2++) {
        // Set the value of the array element.
        // Change this value to the value that the application requires.
        result[(idx0 + 227 * idx1) + 51529 * idx2] = argInit_real_T();
      }
    }
  }
}

//
// Arguments    : void
// Return Type  : real_T
//
static real_T argInit_real_T()
{
  return 0.0;
}

//
// Arguments    : void
// Return Type  : void
//
static void main_alexnet_predict()
{
  real32_T out[1000];
  static real_T b[154587];

  // Initialize function 'alexnet_predict' input arguments.
  // Initialize function input argument 'in'.
  // Call the entry-point 'alexnet_predict'.
  argInit_227x227x3_real_T(b);
  alexnet_predict(b, out);
}

//
// Arguments    : int32_T argc
//                const char * const argv[]
// Return Type  : int32_T
//
int32_T main(int32_T, const char * const [])
{
  // Initialize the application.
  // You do not need to do this more than one time.
  alexnet_predict_initialize();

  // Invoke the entry-point functions.
  // You can call entry-point functions multiple times.
  main_alexnet_predict();

  // Terminate the application.
  // You do not need to do this more than one time.
  alexnet_predict_terminate();
  return 0;
}

//
// File trailer for main.cu
//
// [EOF]
//
