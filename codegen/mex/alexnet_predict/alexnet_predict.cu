/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * alexnet_predict.cu
 *
 * Code generation for function 'alexnet_predict'
 *
 */

/* Include files */
#include "rt_nonfinite.h"
#include "alexnet_predict.h"
#include "DeepLearningNetwork.h"

/* Variable Definitions */
static b_alexnet mynet;
static boolean_T mynet_not_empty;

/* Function Definitions */
void alexnet_predict(const real_T in[154587], real32_T out[1000])
{
  /*  Copyright 2017 The MathWorks, Inc. */
  /*  A persistent object mynet is used to load the series network object. */
  /*  At the first call to this function, the persistent object is constructed and */
  /*  setup. When the function is called subsequent times, the same object is reused  */
  /*  to call predict on inputs, thus avoiding reconstructing and reloading the */
  /*  network object. */
  if (!mynet_not_empty) {
    DeepLearningNetwork_setup(&mynet);
    mynet_not_empty = true;
  }

  /*  pass in input    */
  DeepLearningNetwork_predict(&mynet, in, out);
}

void alexnet_predict_init()
{
  mynet_not_empty = false;
}

/* End of code generation (alexnet_predict.cu) */
