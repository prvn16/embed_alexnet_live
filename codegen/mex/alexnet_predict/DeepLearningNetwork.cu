#include "hip/hip_runtime.h"
/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * DeepLearningNetwork.cu
 *
 * Code generation for function 'DeepLearningNetwork'
 *
 */

/* Include files */
#include "rt_nonfinite.h"
#include "alexnet_predict.h"
#include "DeepLearningNetwork.h"

/* Type Definitions */
#include "cnn_api.hpp"

/* Function Declarations */
static __global__ void c_DeepLearningNetwork_predict_k(const real_T *inputdata,
  real32_T *b_inputdata);
static __global__ void d_DeepLearningNetwork_predict_k(real32_T *inputdata,
  real32_T *inputT);
static __global__ void e_DeepLearningNetwork_predict_k(real32_T *out, real32_T
  *outT);

/* Function Definitions */
static __global__ __launch_bounds__(512, 1) void c_DeepLearningNetwork_predict_k
  (const real_T *inputdata, real32_T *b_inputdata)
{
  int32_T i0;
  ;
  ;
  i0 = (int32_T)(((((gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y)
                    + blockIdx.x) * (blockDim.x * blockDim.y * blockDim.z) +
                   threadIdx.z * blockDim.x * blockDim.y) + threadIdx.y *
                  blockDim.x) + threadIdx.x);
  if (!(int32_T)(i0 >= 154587)) {
    b_inputdata[i0] = (real32_T)inputdata[i0];
  }
}

static __global__ __launch_bounds__(512, 1) void d_DeepLearningNetwork_predict_k
  (real32_T *inputdata, real32_T *inputT)
{
  uint32_T threadId;
  int32_T i0;
  int32_T i1;
  int32_T p;
  ;
  ;
  threadId = ((((gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y) +
                blockIdx.x) * (blockDim.x * blockDim.y * blockDim.z) +
               threadIdx.z * blockDim.x * blockDim.y) + threadIdx.y * blockDim.x)
    + threadIdx.x;
  p = (int32_T)(threadId / 51529U);
  i1 = (int32_T)((threadId - (uint32_T)p * 51529U) / 227U);
  i0 = (int32_T)((threadId - (uint32_T)i1 * 227U) - (uint32_T)p * 51529U);
  if (((int32_T)((!(int32_T)(i0 >= 227)) && (!(int32_T)(i1 >= 227)))) &&
      (!(int32_T)(p >= 3))) {
    inputT[(i0 + 227 * i1) + 51529 * p] = inputdata[(i1 + 227 * i0) + 51529 * p];
  }
}

static __global__ __launch_bounds__(512, 1) void e_DeepLearningNetwork_predict_k
  (real32_T *out, real32_T *outT)
{
  int32_T i0;
  ;
  ;
  i0 = (int32_T)(((((gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y)
                    + blockIdx.x) * (blockDim.x * blockDim.y * blockDim.z) +
                   threadIdx.z * blockDim.x * blockDim.y) + threadIdx.y *
                  blockDim.x) + threadIdx.x);
  if (!(int32_T)(i0 >= 1000)) {
    outT[i0] = out[i0];
  }
}

void DeepLearningNetwork_predict(b_alexnet *obj, const real_T inputdata[154587],
  real32_T outT[1000])
{
  real32_T *gpu_inputT;
  real32_T *gpu_out;
  real_T *gpu_inputdata;
  real32_T *b_gpu_inputdata;
  real32_T *gpu_outT;
  hipMalloc(&gpu_outT, 4000ULL);
  hipMalloc(&gpu_out, 4000ULL);
  hipMalloc(&gpu_inputT, 618348ULL);
  hipMalloc(&b_gpu_inputdata, 618348ULL);
  hipMalloc(&gpu_inputdata, 1236696ULL);
  hipMemcpy((void *)gpu_inputdata, (void *)&inputdata[0], 1236696ULL,
             hipMemcpyHostToDevice);
  c_DeepLearningNetwork_predict_k<<<dim3(302U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (gpu_inputdata, b_gpu_inputdata);
  d_DeepLearningNetwork_predict_k<<<dim3(302U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (b_gpu_inputdata, gpu_inputT);
  hipMemcpy(obj->inputData, gpu_inputT, 154587ULL * sizeof(real32_T),
             hipMemcpyDeviceToDevice);
  obj->predict();
  hipMemcpy(gpu_out, obj->outputData, 1000ULL * sizeof(real32_T),
             hipMemcpyDeviceToDevice);
  e_DeepLearningNetwork_predict_k<<<dim3(2U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (gpu_out, gpu_outT);
  hipMemcpy((void *)&outT[0], (void *)gpu_outT, 4000ULL, hipMemcpyDeviceToHost);
  hipFree(gpu_inputdata);
  hipFree(b_gpu_inputdata);
  hipFree(gpu_inputT);
  hipFree(gpu_out);
  hipFree(gpu_outT);
}

void DeepLearningNetwork_setup(b_alexnet *obj)
{
  obj->setup();
  obj->batchSize = 1;
}

b_alexnet::b_alexnet()
{
  this->numLayers = 25;
  this->cublasHandle = 0;
  this->cudnnHandle = 0;
  this->workSpace = 0;
  this->layers[0] = new MWInputLayer;
  this->layers[1] = new MWConvLayer;
  this->layers[2] = new MWReLULayer;
  this->layers[3] = new MWNormLayer;
  this->layers[4] = new MWMaxPoolingLayer;
  this->layers[5] = new MWConvLayer;
  this->layers[6] = new MWReLULayer;
  this->layers[7] = new MWNormLayer;
  this->layers[8] = new MWMaxPoolingLayer;
  this->layers[9] = new MWConvLayer;
  this->layers[10] = new MWReLULayer;
  this->layers[11] = new MWConvLayer;
  this->layers[12] = new MWReLULayer;
  this->layers[13] = new MWConvLayer;
  this->layers[14] = new MWReLULayer;
  this->layers[15] = new MWMaxPoolingLayer;
  this->layers[16] = new MWFCLayer;
  this->layers[17] = new MWReLULayer;
  this->layers[18] = new MWPassthroughLayer;
  this->layers[19] = new MWFCLayer;
  this->layers[20] = new MWReLULayer;
  this->layers[21] = new MWPassthroughLayer;
  this->layers[22] = new MWFCLayer;
  this->layers[23] = new MWSoftmaxLayer;
  this->layers[24] = new MWOutputLayer;
}

b_alexnet::~b_alexnet()
{
  int32_T idx;
  this->cleanup();
  for (idx = 0; idx < 25; idx++) {
    delete this->layers[idx];
  }
}

void b_alexnet::cleanup()
{
  int32_T idx;
  for (idx = 0; idx < 25; idx++) {
    this->layers[idx]->cleanup();
  }

  if (this->workSpace) {
    hipFree(this->workSpace);
  }

  if (this->cublasHandle) {
    hipblasDestroy(*this->cublasHandle);
  }

  if (this->cudnnHandle) {
    hipdnnDestroy(*this->cudnnHandle);
  }
}

void b_alexnet::predict()
{
  int32_T idx;
  for (idx = 0; idx < 25; idx++) {
    this->layers[idx]->predict();
  }
}

void b_alexnet::setup()
{
  int32_T idx_handles;
  int32_T idx_ws;
  this->cublasHandle = new hipblasHandle_t;
  hipblasCreate(this->cublasHandle);
  this->cudnnHandle = new hipdnnHandle_t;
  hipdnnCreate(this->cudnnHandle);
  for (idx_handles = 0; idx_handles < 25; idx_handles++) {
    this->layers[idx_handles]->setCublasHandle(this->cublasHandle);
    this->layers[idx_handles]->setCudnnHandle(this->cudnnHandle);
  }

  this->layers[0]->createInputLayer(1, 227, 227, 3, 1);
  this->layers[0]->loadAvg(
    "C:\\Sumpurn\\gpucoderdemo_alexnet_live\\codegen\\mex\\alexnet_predict\\cnn_alexnet_avg");
  this->layers[1]->createConvLayer(this->layers[0], 11, 11, 3, 96, 4, 4, 0, 0, 1);
  this->layers[1]->loadWeights(
    "C:\\Sumpurn\\gpucoderdemo_alexnet_live\\codegen\\mex\\alexnet_predict\\cnn_alexnet_conv1_w");
  this->layers[1]->loadBias(
    "C:\\Sumpurn\\gpucoderdemo_alexnet_live\\codegen\\mex\\alexnet_predict\\cnn_alexnet_conv1_b");
  this->layers[2]->createReLULayer(this->layers[1]);
  this->layers[3]->createNormLayer(this->layers[2], 5, 0.0001, 0.75, 1.0);
  this->layers[4]->createMaxPoolingLayer(this->layers[3], 3, 3, 2, 2, 0, 0);
  this->layers[5]->createConvLayer(this->layers[4], 5, 5, 48, 128, 1, 1, 2, 2, 2);
  this->layers[5]->loadWeights(
    "C:\\Sumpurn\\gpucoderdemo_alexnet_live\\codegen\\mex\\alexnet_predict\\cnn_alexnet_conv2_w");
  this->layers[5]->loadBias(
    "C:\\Sumpurn\\gpucoderdemo_alexnet_live\\codegen\\mex\\alexnet_predict\\cnn_alexnet_conv2_b");
  this->layers[6]->createReLULayer(this->layers[5]);
  this->layers[7]->createNormLayer(this->layers[6], 5, 0.0001, 0.75, 1.0);
  this->layers[8]->createMaxPoolingLayer(this->layers[7], 3, 3, 2, 2, 0, 0);
  this->layers[9]->createConvLayer(this->layers[8], 3, 3, 256, 384, 1, 1, 1, 1,
    1);
  this->layers[9]->loadWeights(
    "C:\\Sumpurn\\gpucoderdemo_alexnet_live\\codegen\\mex\\alexnet_predict\\cnn_alexnet_conv3_w");
  this->layers[9]->loadBias(
    "C:\\Sumpurn\\gpucoderdemo_alexnet_live\\codegen\\mex\\alexnet_predict\\cnn_alexnet_conv3_b");
  this->layers[10]->createReLULayer(this->layers[9]);
  this->layers[11]->createConvLayer(this->layers[10], 3, 3, 192, 192, 1, 1, 1, 1,
    2);
  this->layers[11]->loadWeights(
    "C:\\Sumpurn\\gpucoderdemo_alexnet_live\\codegen\\mex\\alexnet_predict\\cnn_alexnet_conv4_w");
  this->layers[11]->loadBias(
    "C:\\Sumpurn\\gpucoderdemo_alexnet_live\\codegen\\mex\\alexnet_predict\\cnn_alexnet_conv4_b");
  this->layers[12]->createReLULayer(this->layers[11]);
  this->layers[13]->createConvLayer(this->layers[12], 3, 3, 192, 128, 1, 1, 1, 1,
    2);
  this->layers[13]->loadWeights(
    "C:\\Sumpurn\\gpucoderdemo_alexnet_live\\codegen\\mex\\alexnet_predict\\cnn_alexnet_conv5_w");
  this->layers[13]->loadBias(
    "C:\\Sumpurn\\gpucoderdemo_alexnet_live\\codegen\\mex\\alexnet_predict\\cnn_alexnet_conv5_b");
  this->layers[14]->createReLULayer(this->layers[13]);
  this->layers[15]->createMaxPoolingLayer(this->layers[14], 3, 3, 2, 2, 0, 0);
  this->layers[16]->createFCLayer(this->layers[15], 9216, 4096);
  this->layers[16]->loadWeights(
    "C:\\Sumpurn\\gpucoderdemo_alexnet_live\\codegen\\mex\\alexnet_predict\\cnn_alexnet_fc6_w");
  this->layers[16]->loadBias(
    "C:\\Sumpurn\\gpucoderdemo_alexnet_live\\codegen\\mex\\alexnet_predict\\cnn_alexnet_fc6_b");
  this->layers[17]->createReLULayer(this->layers[16]);
  this->layers[18]->createPassthroughLayer(this->layers[17]);
  this->layers[19]->createFCLayer(this->layers[18], 4096, 4096);
  this->layers[19]->loadWeights(
    "C:\\Sumpurn\\gpucoderdemo_alexnet_live\\codegen\\mex\\alexnet_predict\\cnn_alexnet_fc7_w");
  this->layers[19]->loadBias(
    "C:\\Sumpurn\\gpucoderdemo_alexnet_live\\codegen\\mex\\alexnet_predict\\cnn_alexnet_fc7_b");
  this->layers[20]->createReLULayer(this->layers[19]);
  this->layers[21]->createPassthroughLayer(this->layers[20]);
  this->layers[22]->createFCLayer(this->layers[21], 4096, 1000);
  this->layers[22]->loadWeights(
    "C:\\Sumpurn\\gpucoderdemo_alexnet_live\\codegen\\mex\\alexnet_predict\\cnn_alexnet_fc8_w");
  this->layers[22]->loadBias(
    "C:\\Sumpurn\\gpucoderdemo_alexnet_live\\codegen\\mex\\alexnet_predict\\cnn_alexnet_fc8_b");
  this->layers[23]->createSoftmaxLayer(this->layers[22]);
  this->layers[24]->createOutputLayer(this->layers[23]);
  this->layers[24]->createWorkSpace((&this->workSpace));
  for (idx_ws = 0; idx_ws < 25; idx_ws++) {
    this->layers[idx_ws]->setWorkSpace(this->workSpace);
  }

  this->inputData = this->layers[0]->getData();
  this->outputData = this->layers[24]->getData();
}

/* End of code generation (DeepLearningNetwork.cu) */
