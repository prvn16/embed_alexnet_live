/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * alexnet_predict_terminate.cu
 *
 * Code generation for function 'alexnet_predict_terminate'
 *
 */

/* Include files */
#include "rt_nonfinite.h"
#include "alexnet_predict.h"
#include "alexnet_predict_terminate.h"
#include "_coder_alexnet_predict_mex.h"
#include "alexnet_predict_data.h"

/* Function Definitions */
void alexnet_predict_atexit()
{
  mexFunctionCreateRootTLS();
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
}

void alexnet_predict_terminate()
{
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
}

/* End of code generation (alexnet_predict_terminate.cu) */
