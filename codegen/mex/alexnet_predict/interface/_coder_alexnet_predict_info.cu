/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * _coder_alexnet_predict_info.cu
 *
 * Code generation for function '_coder_alexnet_predict_info'
 *
 */

/* Include files */
#include "rt_nonfinite.h"
#include "alexnet_predict.h"
#include "_coder_alexnet_predict_info.h"

/* Function Definitions */
mxArray *emlrtMexFcnProperties()
{
  mxArray *xResult;
  mxArray *xEntryPoints;
  const char * fldNames[4] = { "Name", "NumberOfInputs", "NumberOfOutputs",
    "ConstantInputs" };

  mxArray *xInputs;
  const char * b_fldNames[4] = { "Version", "ResolvedFunctions", "EntryPoints",
    "CoverageInfo" };

  xEntryPoints = emlrtCreateStructMatrix(1, 1, 4, fldNames);
  xInputs = emlrtCreateLogicalMatrix(1, 1);
  emlrtSetField(xEntryPoints, 0, "Name", emlrtMxCreateString("alexnet_predict"));
  emlrtSetField(xEntryPoints, 0, "NumberOfInputs", emlrtMxCreateDoubleScalar(1.0));
  emlrtSetField(xEntryPoints, 0, "NumberOfOutputs", emlrtMxCreateDoubleScalar
                (1.0));
  emlrtSetField(xEntryPoints, 0, "ConstantInputs", xInputs);
  xResult = emlrtCreateStructMatrix(1, 1, 4, b_fldNames);
  emlrtSetField(xResult, 0, "Version", emlrtMxCreateString(
    "9.3.0.713579 (R2017b)"));
  emlrtSetField(xResult, 0, "ResolvedFunctions", (mxArray *)
                emlrtMexFcnResolvedFunctionsInfo());
  emlrtSetField(xResult, 0, "EntryPoints", xEntryPoints);
  return xResult;
}

const mxArray *emlrtMexFcnResolvedFunctionsInfo()
{
  const mxArray *nameCaptureInfo;
  const char * data[22] = {
    "789ced5d4d6ce3c615e6b69b4d1745db2041db0445da34288214db5ab6d6b6ec5c6249962c796d4bd68fffd2ad97a246122d724851a42de512f7d0a2bd1441d1"
    "4b811ef658042850e4d200bdb8e8a5c82987b487a228f6d64b81f4d2b34b897ad64f342bdaa2c8a5f406d8a5e9b77cdfccb76fde9b79c3e170b792dbb7388efb",
    "aaf9a7757df227ae5dbe625db8173ad72f70fd65507eab73fdcec03d94e7b8db7dcf81fcb79daba0509d3474eb461229d931e402d1cc1bcacbe44a4d519145ca"
    "533dd75409a791ba229d92625b521225921365b2a5f4dc2444f3468ef788ae6e5aa2d6cfd10a11aa5943e6b44abd5b5da9f786ebe1e731a3fdb76df2b3cae0a7",
    "f5ef2edfebcadf893d8cbe1530eba51a1a0d945543508a442b125939e625d2a0443f96c45312801b55234551d0e7e481fa3e62d4e78ecdfa0e5ea1dce5bed473"
    "37b70678e70c7d76f979958107fc80bc4dc79ca4f0c57542d42dc26b54a4e51da29f295a951bdd7ebbf5b93d70dfadcfed767d840adf32d22bbc8b31f18e9978",
    "56fb41fe4ef2a06d21694d296bbcfc5acb94eb81ed706e2b1c096482f30ba142405714a9a0340275b38ea4186853d6f9fb9e7561f0675a918bf6a3ff27fff7b0"
    "bbf6ea76fff00eafc1d067d71ebfc1c0037b0439d979b0b798e7855221de5c2aed6f14d633ca4abc5b8ff4089c51f5e018f76ee9bf603c8ffdbabff4db616ccd",
    "6d3bb4e282192fe7c4fa866ac4285f905a238471e3c16061d5038a53767338020fe4ced8cd3d93b8402f711e8c27ce7ff9ebb7311ef83d1e94f2c2c9ea02a92f"
    "d68485d34c7843cba4aba7098c07a0ef88a11f7804b9d9afa376fb35cc12ba3f74baf5b050a0f6b7e711a3becedae17fdf06bc9f33f4d9e5ef75061ef007722b",
    "1e88e6e452a3bc3427f3bac417a2e62fcb8426785a943a734ba7e60bcf33ebf57cbb5e06ad52e58c7a30aed8b26f48c3220350181842a1694c2ec687bbff7c88"
    "f1c1eff1a1192ea795cdec41b09c150bb9b87ec097d70fd6313eccf47ce13cee581ee915061ef007722b3e0ccf21b58a5ff3483338bef8f8673fc6b8e0f7b810",
    "cecd879ac9ccf2492991d9db5336c2ebe5e5780ce3c20cf76bae77dec0c2b3cbdf730cbc174c498b3fd210ebad5528affcfee331f1d69878967d80dcb48fed88"
    "5d03b186fc0189a7e5409ba039d9bdffff279797979fb9e8d7670b6f5cbffe75061ed81bc81bfab221e9e9985c5ba68a5259cced8adb210efdfa24fd3a7bbc1f",
    "6b5893f975a2125a24546842a2d7ddf17e777d00f3416ecf1b311f74333cabcc0a1ee6839cd1ff57c6f37679e419fa8147908f39eeeffe74d5bfef09940604a3"
    "4869581561f8efd9f87f5c7b7c9981073c82dc8a139f8f937e8d0bb337be387fef5fbff814e3c284f0dc9a37ec2f49f3bc51499545f16827566bea292d2171d3",
    "13173e613c6f97c713867ee011e4ce8df73693b9aca1aa8aa693e2907eee717e08e711388f18d61e9c47788b87f30867f4e33c6278bbae1b27de67e8b3cbe30f"
    "1978c023c807e2c4d38327e75dbcc071c8d3db67c3be307e4c10cfadf871bf1ebc9fe5a307856029b32806158d4f556ab88f61a279043fad3f9f33f4d9e5efdb",
    "0c3ce00fe44045377498e1730ec2a777fb1970fc31bc5d36ec08df4f9a209e5bf1211a4909dbe57ceea891b9bf20edf30fc26a231dc1f880f1c1193b7c898107"
    "fc81bcbbcfad6aba3722950cdafebd5771e1a31be281febd11782077709fdb1571adb50917ede537fa97310ef83d0e9413d5c2913e1f2a24434b62ac99282fec",
    "2757a3d31307b03f0fbf4261ed5b18d7febec9c003de40def5ff65a2f76c74f6eb3ee76b8c1b9cb2973ee23c78ff8dfe2a8471c0ef7120b5b35bdfcc1c04178d"
    "a55c234c55b576964a61be08e7039c33f9a267651fdb607906e3c2b4d80fe689268887fbd89cd17fc1781efb757f99d43eb651ebbb629dd40c5ef2ccefffe186",
    "78a0ffc1083c90b7d6936d4f08882c0524b100fbd98864fe10e830d59900b8f8bed147fffbc7dfd0cf4f08cf2d3f9fcdd70c7d27b9b797cf647797b55a38b144"
    "785c0f403f3f02ef59f3f3a0ffcec07d17ef4e1bafa81885d6bbade8e7875fa1a09f77070ffdbc33fab11f0faf7fbfddbd7695e75719faecf2c5faff01be406e",
    "5270dc69f1b1a068c42b3f7f3126de43269ed55e90b7ec233d9e81a89a78caeb2430485ddb625cf4fb3ff9f84dcce3f8ddefc7960bab3b27f174733f5eca96f6"
    "63a9d0c26115bf633ac3fdba55de746cbdf77afb91af5e06a5bc5616a950a9fa35afffa3117820bfd678e1290bbe57af815e31670d205c8c072fbd95c1fdc87e",
    "8f07cdc65972958f97ca4bf3422aa6644fe246e80cf3facf6cbf66d9855d3bfc22a33d773bd797fb7efbfa9a757da373fdc19a53e3f551dfad2b498ad2fa909c"
    "5ff34049269e6537201f6ffe583268a0cd9417e724fd39ff06e681fceeff576aa154ae26ad2f552395dcee4656afadaf0b53f4be0ff6e3e1f567e581ce19faec",
    "f2c53a2f0ff802f9c0f85fac470c51d29374c79089260a9ef9fd71df133e60e259ed07f9b5eda5f5e773838441dadcfc8ee9f91f711dc0fffe3f755a8b864b89"
    "d395dcee625388c934183f92a6281f84fe7f78fdfbedee7b8ef9ff6f31f0802f900ff8ffbac04bbc3607a37effeefbca8fc003b933feff9ec55b8fedb8e8ff3f",
    "d87f05fdbfdffd7fa1b29c3ea0925ecc2a72cca089fb8558647303fdffacfafff719faecf2f57d061ef005f201ffcfabaad4ccb69d59dca0822e2a3449d3122f"
    "c051d94ead537f6d44fd405eead4e2b8627dfc0ef0c7fd5e4461043ec89d890f6c5a3dd81f16fde912ae1ffb3d5e6c6d3e100e6be9d5f0e1f6a2560faa62f034",
    "b489dfa3c3fedd2e7353378ff02a8f84f38849e1596556f0701ee18cfe4f18cfdbe55162e8071e413ed1f78abe0bb75677ef6ddf2346fd1db5cbb547aead1f8b"
    "75ca530ed78fedbc7f6c32e5c1bc13f7114c12cfadf30c6a6123252f360f16e7335ad0882e90d55379879b1ebf8ffd7878fd27b57e80ebc72c3c5c3f1e0fcf2a",
    "b38287ebc7cee8bf603c8ffb85fb8b67fb85a91eaceb9a6fbf0bb135020fe4e38c0f4c7e5a0b4c1daadcde27c0bd3887dfff7966fdbcdd71fe61bca89c14e3d4"
    "58ad6c9c448aa9954a723ec7a19f473fef2f3fef553e07fdbce378ed322b78e8e79dd18ffd7878fd27b58e8bf91c161ee673c6c3b3caace0613ec719fde8ff87",
    "d7bfdfee42eeedf325a77ceff9d737c51b2c76f3378f6f8807fad746e081dc9cf76d47ecda43c70e249e96036d82dcf4e34f2e2f2f3f73d18fcf169e6bef6986"
    "d2da6afe2c9d514b4791c38cbe1f2faec4a6e87d9c0bc6f398afe92f5ee56ba8217b9a97ffdd0df1407f74041ec8af737e4b7f78ef30d4310417ed60f383a93e",
    "d7f7fcf7fffe8b77f310d7ce6de4d54af52c533c2c4604b5f4ee6eec5db2793845e7755d309e47ffde5f26754ecbab0c3ce00fe4dd73bbae7237d6b965495975"
    "625c6f37de484a59147abe1bede2b9be4e9ddfd54de37419f4e25cdf930f318feffbf890d9ae8b49a55a4ac4685e95570c636f95cf4dd1f79db17f0f6fd775e3",
    "049eef783dbc1b8c2ff07cc79be1b5cbace0e1f98ee3e9ff3fa7178aa1", "" };

  nameCaptureInfo = NULL;
  emlrtNameCaptureMxArrayR2016a(data, 48120U, &nameCaptureInfo);
  return nameCaptureInfo;
}

/* End of code generation (_coder_alexnet_predict_info.cu) */
