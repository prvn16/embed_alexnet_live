/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * _coder_alexnet_predict_mex.cu
 *
 * Code generation for function '_coder_alexnet_predict_mex'
 *
 */

/* Include files */
#include "alexnet_predict.h"
#include "_coder_alexnet_predict_mex.h"
#include "alexnet_predict_terminate.h"
#include "_coder_alexnet_predict_api.h"
#include "alexnet_predict_initialize.h"
#include "alexnet_predict_data.h"

/* Function Declarations */
static void alexnet_predict_mexFunction(int32_T nlhs, mxArray *plhs[1], int32_T
  nrhs, const mxArray *prhs[1]);

/* Function Definitions */
static void alexnet_predict_mexFunction(int32_T nlhs, mxArray *plhs[1], int32_T
  nrhs, const mxArray *prhs[1])
{
  int32_T n;
  const mxArray *inputs[1];
  const mxArray *outputs[1];
  int32_T b_nlhs;

  /* Check for proper number of arguments. */
  if (nrhs != 1) {
    emlrtErrMsgIdAndTxt(emlrtRootTLSGlobal, "EMLRT:runTime:WrongNumberOfInputs",
                        5, 12, 1, 4, 15, "alexnet_predict");
  }

  if (nlhs > 1) {
    emlrtErrMsgIdAndTxt(emlrtRootTLSGlobal,
                        "EMLRT:runTime:TooManyOutputArguments", 3, 4, 15,
                        "alexnet_predict");
  }

  /* Temporary copy for mex inputs. */
  for (n = 0; n < nrhs; n++) {
    inputs[n] = prhs[n];
  }

  /* Call the function. */
  alexnet_predict_api(inputs, outputs);

  /* Copy over outputs to the caller. */
  if (nlhs < 1) {
    b_nlhs = 1;
  } else {
    b_nlhs = nlhs;
  }

  emlrtReturnArrays(b_nlhs, plhs, outputs);

  /* Module termination. */
  alexnet_predict_terminate();
}

void mexFunction(int32_T nlhs, mxArray *plhs[], int32_T nrhs, const mxArray
                 *prhs[])
{
  mexAtExit(alexnet_predict_atexit);

  /* Initialize the memory manager. */
  /* Module initialization. */
  alexnet_predict_initialize();

  /* Dispatch the entry-point. */
  alexnet_predict_mexFunction(nlhs, plhs, nrhs, prhs);
}

emlrtCTX mexFunctionCreateRootTLS()
{
  emlrtCreateRootTLS(&emlrtRootTLSGlobal, &emlrtContextGlobal, NULL, 1);
  return emlrtRootTLSGlobal;
}

/* End of code generation (_coder_alexnet_predict_mex.cu) */
