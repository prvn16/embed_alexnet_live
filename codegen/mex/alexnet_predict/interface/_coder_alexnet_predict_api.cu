/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * _coder_alexnet_predict_api.cu
 *
 * Code generation for function '_coder_alexnet_predict_api'
 *
 */

/* Include files */
#include "rt_nonfinite.h"
#include "alexnet_predict.h"
#include "_coder_alexnet_predict_api.h"
#include "alexnet_predict_data.h"

/* Function Declarations */
static real_T (*b_emlrt_marshallIn(const mxArray *u, const emlrtMsgIdentifier
  *parentId))[154587];
static real_T (*c_emlrt_marshallIn(const mxArray *src, const emlrtMsgIdentifier *
  msgId))[154587];
static real_T (*emlrt_marshallIn(const mxArray *in, const char_T *identifier))
  [154587];
static const mxArray *emlrt_marshallOut(const real32_T u[1000]);

/* Function Definitions */
static real_T (*b_emlrt_marshallIn(const mxArray *u, const emlrtMsgIdentifier
  *parentId))[154587]
{
  real_T (*y)[154587];
  y = c_emlrt_marshallIn(emlrtAlias(u), parentId);
  emlrtDestroyArray(&u);
  return y;
}
  static real_T (*c_emlrt_marshallIn(const mxArray *src, const
  emlrtMsgIdentifier *msgId))[154587]
{
  real_T (*ret)[154587];
  static const int32_T dims[3] = { 227, 227, 3 };

  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src, "double", false, 3U,
    dims);
  ret = (real_T (*)[154587])emlrtMxGetData(src);
  emlrtDestroyArray(&src);
  return ret;
}

static real_T (*emlrt_marshallIn(const mxArray *in, const char_T *identifier))
  [154587]
{
  real_T (*y)[154587];
  emlrtMsgIdentifier thisId;
  thisId.fIdentifier = const_cast<const char *>(identifier);
  thisId.fParent = NULL;
  thisId.bParentIsCell = false;
  y = b_emlrt_marshallIn(emlrtAlias(in), &thisId);
  emlrtDestroyArray(&in);
  return y;
}
  static const mxArray *emlrt_marshallOut(const real32_T u[1000])
{
  const mxArray *y;
  const mxArray *m0;
  static const int32_T iv0[2] = { 0, 0 };

  static const int32_T iv1[2] = { 1, 1000 };

  y = NULL;
  m0 = emlrtCreateNumericArray(2, iv0, mxSINGLE_CLASS, mxREAL);
  emlrtMxSetData((mxArray *)m0, (void *)&u[0]);
  emlrtSetDimensions((mxArray *)m0, *(int32_T (*)[2])&iv1[0], 2);
  emlrtAssign(&y, m0);
  return y;
}

void alexnet_predict_api(const mxArray * const prhs[1], const mxArray *plhs[1])
{
  real32_T (*out)[1000];
  real_T (*in)[154587];
  out = (real32_T (*)[1000])mxMalloc(sizeof(real32_T [1000]));

  /* Marshall function inputs */
  in = emlrt_marshallIn(emlrtAlias((const mxArray *)prhs[0]), "in");

  /* Invoke the target function */
  alexnet_predict(*in, *out);

  /* Marshall function outputs */
  plhs[0] = emlrt_marshallOut(*out);
}

/* End of code generation (_coder_alexnet_predict_api.cu) */
