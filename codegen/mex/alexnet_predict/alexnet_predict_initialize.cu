/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * alexnet_predict_initialize.cu
 *
 * Code generation for function 'alexnet_predict_initialize'
 *
 */

/* Include files */
#include "rt_nonfinite.h"
#include "alexnet_predict.h"
#include "alexnet_predict_initialize.h"
#include "_coder_alexnet_predict_mex.h"
#include "alexnet_predict_data.h"

/* Function Declarations */
static void alexnet_predict_once();

/* Function Definitions */
static void alexnet_predict_once()
{
  alexnet_predict_init();
}

void alexnet_predict_initialize()
{
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, 0);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  if (emlrtFirstTimeR2012b(emlrtRootTLSGlobal)) {
    alexnet_predict_once();
  }
}

/* End of code generation (alexnet_predict_initialize.cu) */
